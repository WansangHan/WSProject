#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "Calculating.cuh"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "LogManager.h"
#include "SyncServer.h"
#include "PacketManager.h"
#include "WSSockServer.pb.h"

std::unique_ptr<CCalculating> CCalculating::m_inst;
std::once_flag CCalculating::m_once;

CCalculating::CCalculating()
{
}


CCalculating::~CCalculating()
{
	isContinue = false;
	m_calculate_Thread->join();
}

// Object ���� ��ġ ����
__global__ void CalculateCurPosition(ObjectTransform lastMove, ObjectTransform* CurPos, unsigned int _lastTickCount, unsigned int _curTickCount)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	CurPos += tid;
	unsigned int DurTime = _curTickCount - _lastTickCount;
	float MoveTime = DurTime * 0.001f;

	float MoveSpeed = MoveTime* lastMove.m_speed;
	switch (lastMove.m_dir)
	{
	case ObjectDirection::IDLE:
		CurPos->m_vectorX = lastMove.m_vectorX;
		CurPos->m_vectorY = lastMove.m_vectorY;
		break;
	case ObjectDirection::UPUP:
		CurPos->m_vectorX = lastMove.m_vectorX;
		CurPos->m_vectorY = lastMove.m_vectorY - MoveSpeed;
		break;
	case ObjectDirection::LEFT:
		CurPos->m_vectorX = lastMove.m_vectorX - MoveSpeed;
		CurPos->m_vectorY = lastMove.m_vectorY;
		break;
	case ObjectDirection::RGHT:
		CurPos->m_vectorX = lastMove.m_vectorX + MoveSpeed;
		CurPos->m_vectorY = lastMove.m_vectorY;
		break;
	case ObjectDirection::DOWN:
		CurPos->m_vectorX = lastMove.m_vectorX;
		CurPos->m_vectorY = lastMove.m_vectorY + MoveSpeed;
		break;
	case ObjectDirection::UPLE:
		CurPos->m_vectorX = lastMove.m_vectorX - (MoveSpeed * (1 / sqrt(2.0f)));
		CurPos->m_vectorY = lastMove.m_vectorY - (MoveSpeed * (1 / sqrt(2.0f)));
		break;
	case ObjectDirection::UPRG:
		CurPos->m_vectorX = lastMove.m_vectorX + (MoveSpeed * (1 / sqrt(2.0f)));
		CurPos->m_vectorY = lastMove.m_vectorY - (MoveSpeed * (1 / sqrt(2.0f)));
		break;
	case ObjectDirection::DWLE:
		CurPos->m_vectorX = lastMove.m_vectorX - (MoveSpeed * (1 / sqrt(2.0f)));
		CurPos->m_vectorY = lastMove.m_vectorY + (MoveSpeed * (1 / sqrt(2.0f)));
		break;
	case ObjectDirection::DWRG:
		CurPos->m_vectorX = lastMove.m_vectorX + (MoveSpeed * (1 / sqrt(2.0f)));
		CurPos->m_vectorY = lastMove.m_vectorY + (MoveSpeed * (1 / sqrt(2.0f)));
		break;
	}
}

// �浹 ����
__global__ void CalculateCollision(ObjectTransform o1, ObjectTransform o2, bool* iscollision)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	float Distance = pow(o2.m_vectorX - o1.m_vectorX, 2) + pow(o2.m_vectorY - o1.m_vectorY, 2);

	iscollision[tid] = Distance < pow((o2.m_scale * 0.5) - (o1.m_scale * 0.5), 2);
}

// �÷��̾� ���̵�� ��Ī�Ǵ� Player Ŭ���� ������ ã�� �Լ�
std::shared_ptr<CPlayer> CCalculating::FindPlayerToID(int _pID)
{
	tbb::concurrent_hash_map<int, std::shared_ptr<CPlayer>>::accessor a;

	if (m_players.find(a, _pID))
	{
		return a->second;
	}
	else
	{
		CLogManager::getInstance().WriteLogMessage("WARN", true, "Return nullptr in FindPlayer()");
		return nullptr;
	}
}

// ��� ����ó���� ȣ���� �Լ�(������� ȣ��)
void CCalculating::CalculateAll()
{
	while (isContinue)
	{
		// �÷��̾��� ���� ��ǥ�� ���
		for (auto Pr : m_players)
		{
			ObjectTransform* temp;
			hipMalloc((void**)&temp, sizeof(ObjectTransform) * 1);
			hipMemcpy(temp, Pr.second->GetCurTransform().get(), sizeof(ObjectTransform) * 1, hipMemcpyHostToDevice);
			CalculateCurPosition << <1, 1 >> > (*(Pr.second->GetTransform().get()), temp, Pr.second->GetLastGetTickCount(), GetTickCount());
			hipMemcpy(Pr.second->GetCurTransform().get(), temp, sizeof(ObjectTransform) * 1, hipMemcpyDeviceToHost);
			hipFree(temp);
		}

		// �÷��̾ �浹 ����
		for (auto OutterPr : m_players)
		{
			for (auto InnerPr : m_players)
			{
				// �ڱ� �ڽŰ��� �浹 ������ �ƴ϶��
				if (OutterPr.first != InnerPr.first)
				{
					CPlayer smallPlayer;
					CPlayer bigerPlayer;

					// ���� ������Ʈ�� ū ������Ʈ ����
					if (InnerPr.second->GetTransform().get()->m_scale > OutterPr.second->GetTransform().get()->m_scale)
					{
						smallPlayer = *OutterPr.second.get();
						bigerPlayer = *InnerPr.second.get();
					}
					else
					{
						smallPlayer = *InnerPr.second.get();
						bigerPlayer = *OutterPr.second.get();
					}

					bool iscollision = false;
					bool* temp;
					hipMalloc((void**)&temp, sizeof(bool) * 1);
					hipMemcpy(temp, &iscollision, sizeof(bool), hipMemcpyHostToDevice);
					CalculateCollision << <1, 1 >> > (*(smallPlayer.GetCurTransform().get()), *(bigerPlayer.GetCurTransform().get()), temp);
					hipMemcpy(&iscollision, temp, sizeof(bool), hipMemcpyDeviceToHost);
					hipFree(temp);

					// �浹 ��
					if (iscollision)
					{
						WSSockServer::CollisionNotify SendData;
						SendData.set_smallplayerid(smallPlayer.GetID());
						SendData.set_bigerplayerid(bigerPlayer.GetID());
						// Sync �������� �浹�� �Ͼ ������Ʈ�� �˸�
						CSyncServer::getInstance().SendToSyncServer(SendPacketType::SD_COLLISION_NOTIFY, SendData.SerializeAsString(), true);
					}
				}
			}
		}
	}
}

void CCalculating::InitCalculating()
{
	srand((unsigned int)time(NULL));

	isContinue = true;

	m_calculate_Thread = std::unique_ptr<std::thread>(new std::thread([&]() { this->CalculateAll(); }));
}

// ������Ʈ���� ���� ��ġ�� ���ؼ� ����ȭ ������ �����ش�
void CCalculating::SetStartingPosition(std::shared_ptr<CBaseSocket> _sock, sockaddr_in _addr, char* _data, int _size)
{
	WSSockServer::ObjectInformation RecvData;
	RecvData.ParseFromArray(_data, _size);

	float vectorX = rand() % 601;
	float vectorY = rand() % 601;
	float scale = rand() % 50;
	float speed = 20;

	WSSockServer::ObjectPosition* pos = new WSSockServer::ObjectPosition;
	pos->set__id(RecvData._id());
	pos->set__vectorx(vectorX);
	pos->set__vectory(vectorY);
	WSSockServer::ObjectTransform SendData;
	SendData.set_allocated__position(pos);
	SendData.set__scale(scale);
	SendData.set__speed(speed);
	SendData.set__dir((int)ObjectDirection::IDLE);
	CSyncServer::getInstance().SendToSyncServer(SendPacketType::SD_AI_STARTING, SendData.SerializeAsString(), true);
}

// �÷��̾� ���� �� ���� �������� Ŭ���̾�Ʈ ������ ���� �� ���� ��ǥ ����
void CCalculating::EnterPlayer(std::shared_ptr<CBaseSocket> _sock, sockaddr_in _addr, char* _data, int _size)
{
	WSSockServer::PlayerInformation RecvData;
	RecvData.ParseFromArray(_data, _size);

	float vectorX = rand() % 601;
	float vectorY = rand() % 601;
	float scale = rand() % 50;
	float speed = 20;

	std::shared_ptr<CPlayer> player = std::make_shared<CPlayer>();
	player->SetID(RecvData._id());

	ObjectTransform playerTransform(vectorX, vectorY, scale, speed, ObjectDirection::IDLE);
	player->SetTransform(playerTransform);
	player->SetCurTransform(playerTransform);
	player->SetLastGetTickCount(GetTickCount());

	m_players.insert(std::map<int, std::shared_ptr<CPlayer>>::value_type(player->GetID(), player));

	WSSockServer::ObjectPosition* pos = new WSSockServer::ObjectPosition;
	pos->set__id(RecvData._id());
	pos->set__vectorx(vectorX);
	pos->set__vectory(vectorY);
	WSSockServer::ObjectTransform SendData;
	SendData.set_allocated__position(pos);
	SendData.set__scale(scale);
	SendData.set__speed(speed);
	CSyncServer::getInstance().SendToSyncServer(SendPacketType::SD_ENTER_PLAYER_CALC, SendData.SerializeAsString(), true);
}

// �÷��̾� ������ ���� ����
void CCalculating::ApplyPlayerSocket(std::shared_ptr<CBaseSocket> _sock, sockaddr_in _addr, char* _data, int _size)
{
	WSSockServer::ObjectInformation RecvData;
	RecvData.ParseFromArray(_data, _size);

	std::shared_ptr<CPlayer> player = FindPlayerToID(RecvData._id());

	player->SetSocket(_sock);

	CPacketManager::getInstance().SendPacketToServer(player->GetSocket(), SendPacketType::SD_SUCCESS_CALC_TCP, "", nullptr, true);
}

// �÷��̾� ������ UDP ��巹�� ����
void CCalculating::ApplyPlayerUDP(std::shared_ptr<CBaseSocket> _sock, sockaddr_in _addr, char * _data, int _size)
{
	WSSockServer::ObjectInformation RecvData;
	RecvData.ParseFromArray(_data, _size);

	std::shared_ptr<CPlayer> player = FindPlayerToID(RecvData._id());
	player->SetAddr(_addr);

	CPacketManager::getInstance().SendPacketToServer(player->GetSocket(), SendPacketType::SD_SUCCESS_CALC_UDP, "", nullptr, true);
}

// �÷��̾ ������ ��, �ش� ���̵� �´� �÷��̾� ����
void CCalculating::ExitPlayer(std::shared_ptr<CBaseSocket> _sock, sockaddr_in _addr, char* _data, int _size)
{
	WSSockServer::ObjectInformation RecvData;
	RecvData.ParseFromArray(_data, _size);

	m_players.erase(RecvData._id());
}

// Ŭ���̾�Ʈ�� ������ �̵� �̺�Ʈ ������ ����
void CCalculating::ApplyPlayerTrasform(std::shared_ptr<CBaseSocket> _sock, sockaddr_in _addr, char * _data, int _size)
{
	WSSockServer::ObjectTransform RecvData;
	RecvData.ParseFromArray(_data, _size);

	std::shared_ptr<CPlayer> player = FindPlayerToID(RecvData._position()._id());
	player->SetLastGetTickCount(GetTickCount());
	ObjectTransform playerTransform(RecvData._position()._vectorx(), RecvData._position()._vectory(), RecvData._scale(), RecvData._speed(), (ObjectDirection)RecvData._dir());
	player->SetTransform(playerTransform);
	player->SetCurTransform(playerTransform);
}

// ������ �÷��̾��� ũ�� ����
void CCalculating::IncreaseScale(std::shared_ptr<CBaseSocket> _sock, sockaddr_in _addr, char * _data, int _size)
{
	WSSockServer::IncreaseScale RecvData;
	RecvData.ParseFromArray(_data, _size);

	std::shared_ptr<CPlayer> player = FindPlayerToID(RecvData._id());

	std::shared_ptr<ObjectTransform> temp = player->GetTransform();
	temp->m_scale = RecvData._increase();
	player->SetTransform(*temp.get());
}
